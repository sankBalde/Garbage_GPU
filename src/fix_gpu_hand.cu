#include "fix_gpu_hand.cuh"
#include "utils.cuh"
#include <raft/common/nvtx.hpp>


bool check_predicate(const rmm::device_uvector<int>& predicate_gpu, const std::vector<int>& predicate_cpu) {
    // Vérifier que les tailles des deux vecteurs sont identiques
    if (predicate_gpu.size() != predicate_cpu.size()) {
        std::cerr << "Size mismatch: GPU predicate size (" << predicate_gpu.size() 
                  << ") vs CPU predicate size (" << predicate_cpu.size() << ")" << std::endl;
        return false;
    }

    // Créer un vecteur hôte pour recevoir les données du GPU
    std::vector<int> predicate_gpu_host(predicate_gpu.size());

    // Copier les données du GPU vers le vecteur hôte
    CUDA_CHECK_ERROR(hipMemcpy(predicate_gpu_host.data(), predicate_gpu.data(), 
                                predicate_gpu.size() * sizeof(int), hipMemcpyDeviceToHost));

    // Comparer les deux vecteurs élément par élément
    for (std::size_t i = 0; i < predicate_cpu.size(); ++i) {
        if (predicate_gpu_host[i] != predicate_cpu[i]) {
            std::cerr << "Mismatch at index " << i << ": CPU(" << predicate_cpu[i] 
                      << ") vs GPU(" << predicate_gpu_host[i] << ")" << std::endl;
            return false;
        }
    }

    // Si toutes les valeurs sont identiques, les vecteurs sont identiques
    return true;
}



void fix_image_gpu_hand_old(Image& to_fix)
{
    const int image_size = to_fix.width * to_fix.height;
    int block_size = 256;
    int grid_size_non_garbage = (image_size + block_size - 1) / block_size;
    int grid_size_avec_garbage = (to_fix.size() + block_size - 1) / block_size;

    // Allocation sur GPU pour d_buffer
    rmm::device_uvector<int> d_buffer(to_fix.size(), rmm::cuda_stream_default);

    // Si to_fix.buffer est en mémoire hôte (CPU), il faut le copier sur le GPU.
    // Si to_fix.buffer est déjà sur le GPU, tu peux le faire directement.
    CUDA_CHECK_ERROR(hipMemcpy(d_buffer.data(), to_fix.buffer, to_fix.size() * sizeof(int), hipMemcpyHostToDevice));

    // Allocation sur GPU pour predicate
    rmm::device_uvector<int> predicate(to_fix.size(), d_buffer.stream());
    
    //TODO: Pas besoin ?
    // CUDA_CHECK_ERROR(hipMemset(predicate.data(), 0, predicate.size() * sizeof(int)));

    constexpr int garbage_val = -27;

    // Lancement du kernel avec les données sur GPU
    build_predicate_kernel<<<grid_size_avec_garbage, block_size, 0, d_buffer.stream()>>>(
        raft::device_span<int>(d_buffer.data(), d_buffer.size()),
        raft::device_span<int>(predicate.data(), predicate.size()),
        garbage_val, to_fix.size());
    CUDA_CHECK_ERROR(hipGetLastError());


    CUDA_CHECK_ERROR(hipStreamSynchronize(d_buffer.stream()));

    /*std::vector<int> predicate_CPU(to_fix.size(), 0);

    for (int i = 0; i < to_fix.size(); ++i) {
        if (to_fix.buffer[i] != garbage_val)
            predicate_CPU[i] = 1;
    }*/



    // Synchronisation pour assurer la fin de l'exécution

    // Appel de your_scan pour effectuer un scan exclusif
    your_scan(predicate, true);
    //std::inclusive_scan(predicate_CPU.begin(), predicate_CPU.end(), predicate_CPU.begin(), 0);

    //check_predicate(predicate, predicate_CPU);

    //TODO: Pas besoin ? Deja un streamSynchronize dans your_scan
    CUDA_CHECK_ERROR(hipStreamSynchronize(d_buffer.stream()));

    // Lancement du kernel de scatter
    scatter_kernel<<<grid_size_avec_garbage, block_size, 0, d_buffer.stream()>>>(
        raft::device_span<int>(d_buffer.data(), d_buffer.size()),
        raft::device_span<int>(predicate.data(), predicate.size()),
        to_fix.size());

    CUDA_CHECK_ERROR(hipStreamSynchronize(d_buffer.stream()));

     apply_map_kernel<<<grid_size_non_garbage, block_size, 0, d_buffer.stream()>>>(
        raft::device_span<int>(d_buffer.data(), d_buffer.size()),
         image_size);
    CUDA_CHECK_ERROR(hipStreamSynchronize(d_buffer.stream()));



    //! Mettre le d_buffer dans le to_fix -> Format CPU
    //CUDA_CHECK_ERROR(hipMemcpy(to_fix.buffer, d_buffer.data(), image_size * sizeof(int), hipMemcpyDeviceToHost));


    // #3 Histogram equalization

    // Histogram


    //! GPU

    // // Allocation pour l'histogramme et le CDF
    rmm::device_uvector<int> histogram(256, d_buffer.stream());
    rmm::device_uvector<int> cdf(256, rmm::cuda_stream_default);

    //CUDA_CHECK_ERROR(hipMemset(histogram.data(), 0, histogram.size() * sizeof(int)));
    //CUDA_CHECK_ERROR(hipMemset(cdf.data(), 0, cdf.size() * sizeof(int)));

    // // Lancement du kernel pour calculer l'histogramme
     histogram_kernel<<<grid_size_avec_garbage, block_size, 0, d_buffer.stream()>>>(
         raft::device_span<int>(d_buffer.data(), d_buffer.size()),
         raft::device_span<int>(histogram.data(), histogram.size()),
         image_size);

    //TODO: Besoin des 2 ?
    CUDA_CHECK_ERROR(hipStreamSynchronize(histogram.stream()));
    CUDA_CHECK_ERROR(hipStreamSynchronize(d_buffer.stream()));

    your_scan(histogram, false);

    //TODO: Pas besoin ? Deja un streamSynchronize dans your_scan + plutot histogram.stream() ?
    CUDA_CHECK_ERROR(hipStreamSynchronize(d_buffer.stream()));

    // // Trouver le premier élément non nul dans le CDF
    //TODO: Ne pas utiliser un host ? (Petit kernel 256x1)
    std::vector<int> histogram_host(256);
    CUDA_CHECK_ERROR(hipMemcpy(histogram_host.data(), histogram.data(), histogram.size() * sizeof(int), hipMemcpyDeviceToHost));

    // // Trouver le premier élément non nul dans l'histogramme
     int cdf_min = 0;
     for (int i = 1; i < 256; ++i)
     {
         if (histogram_host[i] != 0)
         {
             cdf_min = histogram_host[i];
             break;
         }
     }

    // // Appliquer l'égalisation de l'histogramme
     equalize_kernel<<<grid_size_non_garbage, block_size, 0, d_buffer.stream()>>>(
         raft::device_span<int>(d_buffer.data(), d_buffer.size()),
         raft::device_span<int>(d_buffer.data(), d_buffer.size()),  // Réutilisation de d_buffer pour stocker le résultat
         raft::device_span<int>(histogram.data(), histogram.size()),
         cdf_min, image_size);

    // // Synchronisation pour assurer la fin de l'exécution
   CUDA_CHECK_ERROR(hipStreamSynchronize(d_buffer.stream()));
   CUDA_CHECK_ERROR(hipMemcpy(to_fix.buffer, d_buffer.data(), image_size * sizeof(int), hipMemcpyDeviceToHost));
}


void fix_image_gpu_hand(Image& to_fix)
{
    raft::common::nvtx::range fun_scope("Fix Image GPU Hand");
    const int image_size = to_fix.width * to_fix.height;
    int block_size = 256;
    int grid_size_non_garbage = (image_size + block_size - 1) / block_size;
    int grid_size_avec_garbage = (to_fix.size() + block_size - 1) / block_size;

    rmm::device_uvector<int> d_buffer(to_fix.size(), rmm::cuda_stream_default);

    // Copie to_fix sur GPU
    CUDA_CHECK_ERROR(hipMemcpy(d_buffer.data(), to_fix.buffer, to_fix.size() * sizeof(int), hipMemcpyHostToDevice));

    rmm::device_uvector<int> predicate(to_fix.size(), d_buffer.stream());
    
//! 1 - GARBAGE VALUES (-27)

    // Initialiser predicate à zéro
    CUDA_CHECK_ERROR(hipMemset(predicate.data(), 0, predicate.size() * sizeof(int)));

    constexpr int garbage_val = -27;
    raft::common::nvtx::push_range("Build Predicate Kernel");
    build_predicate_kernel<<<grid_size_avec_garbage, block_size, 0, d_buffer.stream()>>>(
        raft::device_span<int>(d_buffer.data(), d_buffer.size()),
        raft::device_span<int>(predicate.data(), predicate.size()),
        garbage_val, to_fix.size());
    CUDA_CHECK_ERROR(hipGetLastError());
    CUDA_CHECK_ERROR(hipStreamSynchronize(d_buffer.stream()));
    raft::common::nvtx::pop_range();

    // Scan exclusif
    raft::common::nvtx::push_range("Scan Exclusif Predicate");
    your_scan(predicate, true);
    raft::common::nvtx::pop_range();

    //? Rajouter la ligne en dessous pour le nouveau scatter_kernel
    // int shared_mem_size = block_size * sizeof(int);
    raft::common::nvtx::push_range("Scatter Kernel");
    scatter_kernel<<<grid_size_avec_garbage, block_size, 0, d_buffer.stream()>>>(
        raft::device_span<int>(d_buffer.data(), d_buffer.size()),
        raft::device_span<int>(predicate.data(), predicate.size()),
        to_fix.size());
    CUDA_CHECK_ERROR(hipStreamSynchronize(d_buffer.stream()));
    raft::common::nvtx::pop_range();

//! 2 - MAP
    raft::common::nvtx::push_range("Apply Map Kernel");
    apply_map_kernel<<<grid_size_non_garbage, block_size, 0, d_buffer.stream()>>>(
        raft::device_span<int>(d_buffer.data(), d_buffer.size()),
        image_size);
    CUDA_CHECK_ERROR(hipStreamSynchronize(d_buffer.stream()));
    raft::common::nvtx::pop_range();

//! 3 - HISTOGRAM EGALISATION

    rmm::device_uvector<int> histogram(256, d_buffer.stream());
    rmm::device_uvector<int> cdf(256, rmm::cuda_stream_default);

    // Initialise

    CUDA_CHECK_ERROR(hipMemset(histogram.data(), 0, histogram.size() * sizeof(int)));
    const int histogram_sharedMem = 256 * sizeof(int);

    raft::common::nvtx::push_range("Histogram Kernel");
    histogram_kernel<<<grid_size_avec_garbage, block_size, histogram_sharedMem, d_buffer.stream()>>>(
        raft::device_span<int>(d_buffer.data(), d_buffer.size()),
        raft::device_span<int>(histogram.data(), histogram.size()),
        image_size);

    CUDA_CHECK_ERROR(hipStreamSynchronize(d_buffer.stream()));
    raft::common::nvtx::pop_range();

    raft::common::nvtx::push_range("Scan Inclusif Kernel");
    your_scan(histogram, false);    // Scan inclusif
    raft::common::nvtx::pop_range();

    //TODO: Ne pas utiliser un host ? (-> Petit kernel 256x1)
    std::vector<int> histogram_host(256);
    CUDA_CHECK_ERROR(hipMemcpy(histogram_host.data(), histogram.data(), histogram.size() * sizeof(int), hipMemcpyDeviceToHost));

    // int cdf_min = 0;
    
    // for (int i = 1; i < 256; ++i)
    // {
    //     if (histogram_host[i] != 0)
    //     {
    //         cdf_min = histogram_host[i];
    //         break;
    //     }
    // }
    auto first_none_zero = std::find_if(histogram_host.begin(), histogram_host.end(), [](auto v) { return v != 0; });

    const int cdf_min = *first_none_zero;

    raft::common::nvtx::push_range("Equalize Kernel");
    equalize_kernel<<<grid_size_non_garbage, block_size, 0, d_buffer.stream()>>>(
        raft::device_span<int>(d_buffer.data(), d_buffer.size()),
        raft::device_span<int>(d_buffer.data(), d_buffer.size()),  // Réutilisation de d_buffer pour stocker le résultat
        raft::device_span<int>(histogram.data(), histogram.size()),
        cdf_min, image_size);

    CUDA_CHECK_ERROR(hipStreamSynchronize(d_buffer.stream()));
    raft::common::nvtx::pop_range();
    CUDA_CHECK_ERROR(hipMemcpy(to_fix.buffer, d_buffer.data(), image_size * sizeof(int), hipMemcpyDeviceToHost));
}


void your_reduce(rmm::device_uvector<int>& buffer,
                 rmm::device_scalar<int>& total)
{
    constexpr int blocksize = 256;
    int gridsize = (buffer.size() + blocksize - 1) / blocksize;

    int shared_memory_size = blocksize * sizeof(int);

    if (gridsize == 1) {
        kernel_your_reduce_grid_stride_loop<int><<<gridsize, blocksize, shared_memory_size, buffer.stream()>>>(
            raft::device_span<const int>(buffer.data(), buffer.size()),
            raft::device_span<int>(total.data(), 1));
        CUDA_CHECK_ERROR(hipStreamSynchronize(buffer.stream()));
        return;
    }

//! Si on a plus de 64 valeurs

    rmm::device_uvector<int> partial_sums(gridsize, buffer.stream());
    rmm::device_uvector<int> partial_sums_bis(gridsize, buffer.stream());

    kernel_your_reduce_grid_stride_loop<int><<<gridsize, blocksize, shared_memory_size, buffer.stream()>>>(
        raft::device_span<const int>(buffer.data(), buffer.size()),
        raft::device_span<int>(partial_sums.data(), partial_sums.size()));
    CUDA_CHECK_ERROR(hipStreamSynchronize(buffer.stream()));

    int cpt = 0;
    
    while (gridsize > 1) {

        gridsize = (gridsize + blocksize - 1) / blocksize;

        if (cpt % 2 == 0) {
            kernel_your_reduce_grid_stride_loop<int><<<gridsize, blocksize, shared_memory_size, buffer.stream()>>>(
                raft::device_span<const int>(partial_sums.data(), partial_sums.size()),
                raft::device_span<int>(partial_sums_bis.data(), partial_sums_bis.size()));
            CUDA_CHECK_ERROR(hipStreamSynchronize(buffer.stream()));

            partial_sums_bis.resize(gridsize, buffer.stream());
        } else {
            kernel_your_reduce_grid_stride_loop<int><<<gridsize, blocksize, shared_memory_size, buffer.stream()>>>(
                raft::device_span<const int>(partial_sums_bis.data(), partial_sums_bis.size()),
                raft::device_span<int>(partial_sums.data(), partial_sums.size()));
            CUDA_CHECK_ERROR(hipStreamSynchronize(buffer.stream()));

            partial_sums.resize(gridsize, buffer.stream());
        }

        //? Retire pour les performances
        // partial_sums = std::move(new_partial_sums);

        cpt += 1;
    }

    kernel_your_reduce_grid_stride_loop<int><<<1, blocksize, shared_memory_size, buffer.stream()>>>(
        raft::device_span<int>(partial_sums.data(), partial_sums.size()),
        raft::device_span<int>(total.data(), 1));
    CUDA_CHECK_ERROR(hipStreamSynchronize(buffer.stream()));
}