#include "fix_gpu_hand.cuh"
#include "utils.cuh"
#include <raft/common/nvtx.hpp>

void fix_image_gpu_hand_old(Image& to_fix)
{
    const int image_size = to_fix.width * to_fix.height;
    int block_size = 256;
    int grid_size_non_garbage = (image_size + block_size - 1) / block_size;
    int grid_size_avec_garbage = (to_fix.size() + block_size - 1) / block_size;

    // Allocation sur GPU pour d_buffer
    rmm::device_uvector<int> d_buffer(to_fix.size(), rmm::cuda_stream_default);

    CUDA_CHECK_ERROR(hipMemcpy(d_buffer.data(), to_fix.buffer, to_fix.size() * sizeof(int), hipMemcpyHostToDevice));

    // Allocation sur GPU pour predicate
    rmm::device_uvector<int> predicate(to_fix.size(), d_buffer.stream());

    constexpr int garbage_val = -27;

    // Lancement du kernel avec les données sur GPU
    build_predicate_kernel<<<grid_size_avec_garbage, block_size, 0, d_buffer.stream()>>>(
        raft::device_span<int>(d_buffer.data(), d_buffer.size()),
        raft::device_span<int>(predicate.data(), predicate.size()),
        garbage_val, to_fix.size());
    CUDA_CHECK_ERROR(hipGetLastError());


    CUDA_CHECK_ERROR(hipStreamSynchronize(d_buffer.stream()));

    // Appel de your_scan pour effectuer un scan exclusif
    your_scan(predicate, true);

    CUDA_CHECK_ERROR(hipStreamSynchronize(d_buffer.stream()));

    // Lancement du kernel de scatter
    scatter_kernel<<<grid_size_avec_garbage, block_size, 0, d_buffer.stream()>>>(
        raft::device_span<int>(d_buffer.data(), d_buffer.size()),
        raft::device_span<int>(predicate.data(), predicate.size()),
        to_fix.size());

    CUDA_CHECK_ERROR(hipStreamSynchronize(d_buffer.stream()));

     apply_map_kernel<<<grid_size_non_garbage, block_size, 0, d_buffer.stream()>>>(
        raft::device_span<int>(d_buffer.data(), d_buffer.size()),
         image_size);
    CUDA_CHECK_ERROR(hipStreamSynchronize(d_buffer.stream()));

    // #3 Histogram equalization

    // Histogram

    //! GPU

    // // Allocation pour l'histogramme et le CDF
    rmm::device_uvector<int> histogram(256, d_buffer.stream());
    rmm::device_uvector<int> cdf(256, rmm::cuda_stream_default);

    // // Lancement du kernel pour calculer l'histogramme
     histogram_kernel<<<grid_size_avec_garbage, block_size, 0, d_buffer.stream()>>>(
         raft::device_span<int>(d_buffer.data(), d_buffer.size()),
         raft::device_span<int>(histogram.data(), histogram.size()),
         image_size);

    CUDA_CHECK_ERROR(hipStreamSynchronize(histogram.stream()));
    CUDA_CHECK_ERROR(hipStreamSynchronize(d_buffer.stream()));

    your_scan(histogram, false);

    CUDA_CHECK_ERROR(hipStreamSynchronize(d_buffer.stream()));

    std::vector<int> histogram_host(256);
    CUDA_CHECK_ERROR(hipMemcpy(histogram_host.data(), histogram.data(), histogram.size() * sizeof(int), hipMemcpyDeviceToHost));

    // // Trouver le premier élément non nul dans l'histogramme
     int cdf_min = 0;
     for (int i = 1; i < 256; ++i)
     {
         if (histogram_host[i] != 0)
         {
             cdf_min = histogram_host[i];
             break;
         }
     }

    // // Appliquer l'égalisation de l'histogramme
     equalize_kernel<<<grid_size_non_garbage, block_size, 0, d_buffer.stream()>>>(
         raft::device_span<int>(d_buffer.data(), d_buffer.size()),
         raft::device_span<int>(d_buffer.data(), d_buffer.size()),  // Réutilisation de d_buffer pour stocker le résultat
         raft::device_span<int>(histogram.data(), histogram.size()),
         cdf_min, image_size);

    // // Synchronisation pour assurer la fin de l'exécution
   CUDA_CHECK_ERROR(hipStreamSynchronize(d_buffer.stream()));
   CUDA_CHECK_ERROR(hipMemcpy(to_fix.buffer, d_buffer.data(), image_size * sizeof(int), hipMemcpyDeviceToHost));
}


void fix_image_gpu_hand(Image& to_fix)
{
    raft::common::nvtx::range fun_scope("Fix Image GPU Hand");
    const int image_size = to_fix.width * to_fix.height;
    int block_size = 256;
    int grid_size_non_garbage = (image_size + block_size - 1) / block_size;
    int grid_size_avec_garbage = (to_fix.size() + block_size - 1) / block_size;

    rmm::device_uvector<int> d_buffer(to_fix.size(), rmm::cuda_stream_default);

    // Copie to_fix sur GPU
    CUDA_CHECK_ERROR(hipMemcpy(d_buffer.data(), to_fix.buffer, to_fix.size() * sizeof(int), hipMemcpyHostToDevice));

    rmm::device_uvector<int> predicate(to_fix.size(), d_buffer.stream());
    
//! 1 - GARBAGE VALUES (-27)

    // Initialiser predicate à zéro
    CUDA_CHECK_ERROR(hipMemset(predicate.data(), 0, predicate.size() * sizeof(int)));

    constexpr int garbage_val = -27;
    raft::common::nvtx::push_range("Build Predicate Kernel");
    build_predicate_kernel<<<grid_size_avec_garbage, block_size, 0, d_buffer.stream()>>>(
        raft::device_span<int>(d_buffer.data(), d_buffer.size()),
        raft::device_span<int>(predicate.data(), predicate.size()),
        garbage_val, to_fix.size());
    CUDA_CHECK_ERROR(hipGetLastError());
    CUDA_CHECK_ERROR(hipStreamSynchronize(d_buffer.stream()));
    raft::common::nvtx::pop_range();

    // Scan exclusif
    raft::common::nvtx::push_range("Scan Exclusif Predicate");
    your_scan(predicate, true);
    raft::common::nvtx::pop_range();

    raft::common::nvtx::push_range("Scatter Kernel");
    scatter_kernel<<<grid_size_avec_garbage, block_size, 0, d_buffer.stream()>>>(
        raft::device_span<int>(d_buffer.data(), d_buffer.size()),
        raft::device_span<int>(predicate.data(), predicate.size()),
        to_fix.size());
    CUDA_CHECK_ERROR(hipStreamSynchronize(d_buffer.stream()));
    raft::common::nvtx::pop_range();

//! 2 - MAP
    raft::common::nvtx::push_range("Apply Map Kernel");
    apply_map_kernel<<<grid_size_non_garbage, block_size, 0, d_buffer.stream()>>>(
        raft::device_span<int>(d_buffer.data(), d_buffer.size()),
        image_size);
    CUDA_CHECK_ERROR(hipStreamSynchronize(d_buffer.stream()));
    raft::common::nvtx::pop_range();

//! 3 - HISTOGRAM EGALISATION

    rmm::device_uvector<int> histogram(256, d_buffer.stream());
    rmm::device_uvector<int> cdf(256, rmm::cuda_stream_default);

    // Initialise

    CUDA_CHECK_ERROR(hipMemset(histogram.data(), 0, histogram.size() * sizeof(int)));
    const int histogram_sharedMem = 256 * sizeof(int);

    raft::common::nvtx::push_range("Histogram Kernel");
    histogram_kernel<<<grid_size_avec_garbage, block_size, histogram_sharedMem, d_buffer.stream()>>>(
        raft::device_span<int>(d_buffer.data(), d_buffer.size()),
        raft::device_span<int>(histogram.data(), histogram.size()),
        image_size);

    CUDA_CHECK_ERROR(hipStreamSynchronize(d_buffer.stream()));
    raft::common::nvtx::pop_range();

    raft::common::nvtx::push_range("Scan Inclusif Kernel");
    your_scan(histogram, false);    // Scan inclusif
    raft::common::nvtx::pop_range();

    std::vector<int> histogram_host(256);
    CUDA_CHECK_ERROR(hipMemcpy(histogram_host.data(), histogram.data(), histogram.size() * sizeof(int), hipMemcpyDeviceToHost));

    auto first_none_zero = std::find_if(histogram_host.begin(), histogram_host.end(), [](auto v) { return v != 0; });

    const int cdf_min = *first_none_zero;

    raft::common::nvtx::push_range("Equalize Kernel");
    equalize_kernel<<<grid_size_non_garbage, block_size, 0, d_buffer.stream()>>>(
        raft::device_span<int>(d_buffer.data(), d_buffer.size()),
        raft::device_span<int>(d_buffer.data(), d_buffer.size()),
        raft::device_span<int>(histogram.data(), histogram.size()),
        cdf_min, image_size);

    CUDA_CHECK_ERROR(hipStreamSynchronize(d_buffer.stream()));
    raft::common::nvtx::pop_range();
    CUDA_CHECK_ERROR(hipMemcpy(to_fix.buffer, d_buffer.data(), image_size * sizeof(int), hipMemcpyDeviceToHost));
}


void your_reduce(rmm::device_uvector<int>& buffer,
                 rmm::device_scalar<int>& total)
{
    constexpr int blocksize = 256;
    int gridsize = (buffer.size() + blocksize - 1) / blocksize;

    int shared_memory_size = blocksize * sizeof(int);

    if (gridsize == 1) {
        kernel_your_reduce_grid_stride_loop<int><<<gridsize, blocksize, shared_memory_size, buffer.stream()>>>(
            raft::device_span<const int>(buffer.data(), buffer.size()),
            raft::device_span<int>(total.data(), 1));
        CUDA_CHECK_ERROR(hipStreamSynchronize(buffer.stream()));
        return;
    }

//! Si on a plus de 64 valeurs

    rmm::device_uvector<int> partial_sums(gridsize, buffer.stream());
    rmm::device_uvector<int> partial_sums_bis(gridsize, buffer.stream());

    kernel_your_reduce_grid_stride_loop<int><<<gridsize, blocksize, shared_memory_size, buffer.stream()>>>(
        raft::device_span<const int>(buffer.data(), buffer.size()),
        raft::device_span<int>(partial_sums.data(), partial_sums.size()));
    CUDA_CHECK_ERROR(hipStreamSynchronize(buffer.stream()));

    int cpt = 0;
    
    while (gridsize > 1) {

        gridsize = (gridsize + blocksize - 1) / blocksize;

        if (cpt % 2 == 0) {
            kernel_your_reduce_grid_stride_loop<int><<<gridsize, blocksize, shared_memory_size, buffer.stream()>>>(
                raft::device_span<const int>(partial_sums.data(), partial_sums.size()),
                raft::device_span<int>(partial_sums_bis.data(), partial_sums_bis.size()));
            CUDA_CHECK_ERROR(hipStreamSynchronize(buffer.stream()));

            partial_sums_bis.resize(gridsize, buffer.stream());
        } else {
            kernel_your_reduce_grid_stride_loop<int><<<gridsize, blocksize, shared_memory_size, buffer.stream()>>>(
                raft::device_span<const int>(partial_sums_bis.data(), partial_sums_bis.size()),
                raft::device_span<int>(partial_sums.data(), partial_sums.size()));
            CUDA_CHECK_ERROR(hipStreamSynchronize(buffer.stream()));

            partial_sums.resize(gridsize, buffer.stream());
        }

        cpt += 1;
    }

    kernel_your_reduce_grid_stride_loop<int><<<1, blocksize, shared_memory_size, buffer.stream()>>>(
        raft::device_span<int>(partial_sums.data(), partial_sums.size()),
        raft::device_span<int>(total.data(), 1));
    CUDA_CHECK_ERROR(hipStreamSynchronize(buffer.stream()));
}